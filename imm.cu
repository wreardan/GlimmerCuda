#include "hip/hip_runtime.h"
#include "imm.h"

#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <algorithm>
#include <sstream>

using namespace std;


static string bases = "ACGT";
//convert bases to indices
//i.e. "ACGTGCA" -> 0123210
string bases_to_indices(string sequence) {
	string result;
	for(char c : sequence) {
		int index = bases.find(c);
		//TODO: something more reasonable here?
		if(index == string::npos) {
			index = rand() % bases.size();
		}

		result.push_back(index);
	}
	return result;
}


IMM::IMM() {
	order = -1;
	window = -1;
	d_counts = NULL;
}


void IMM::dispose() {
	if(d_counts != NULL) {
		hipFree(d_counts);
		d_counts = NULL;
		order = -1;
	}
}


IMM::~IMM() {
	dispose();
}


//Take x to power y
//http://www.bedroomlan.org/writing/oric-c-programming/iteration-vs-recursion
__host__ __device__ unsigned int power (unsigned int x, unsigned int y)
{
	unsigned int result;

	if (y == 0) {
		return 1;
	}

	for (result = x; y > 1; y--) {
		result *= x;
	}

	return result;
}


//window is the size of the sliding window
//order is the markov-order of the model
void IMM::init(int window, int order) {
	this->order = order;
	this->window = window;

	order_sum = 0;
	for(int i = 0; i <= order; i++) {
		order_sum += power(4, i+1);
	}

	total_bytes = window * order_sum * sizeof(int);
	
    hipError_t cudaStatus;
	cudaStatus = hipMalloc(&d_counts, total_bytes);
	assert (hipSuccess == cudaStatus);
	cudaStatus = hipMemset(d_counts, 0, total_bytes);
	assert (hipSuccess == cudaStatus);
}

__host__ __device__ int get_sequence_index(char * sequence, int length) {
	int index = 0;
	for(int i = 0; i < length; i++) {
		index = index * 4 + *(sequence+i);
	}
	return index;
}



//this kernel builds the imm from a set of training sequences
__global__ void counting_kernel(int *model, char * sequences, int pos_size, int max_order, int window) {
    int num = threadIdx.x; //sequence number
	int position = threadIdx.y; //position index
	int order = threadIdx.z; //order number

	if(position + order >= window) {
		return;
	}
    
	//get sequence
	char * sequence = sequences + num * window + position;

	//compute index, order_index
	int index = get_sequence_index(sequence, order+1);
	int order_index = 0;
	for(int i = 0; i < order; i++) {
		order_index += power(4, i+1);
	}

	//increment count
	int * count = model + index + order_index + pos_size * order;
	count += position * pos_size;
	atomicAdd(count, 1);
}


void send_windows_to_gpu(vector<string> & sequences, int window, char **d_seq) {
    hipError_t cudaStatus;
	//Concatenate sequences
    stringstream ss;
	for(string & seq : sequences) {
		ss << seq.substr(0, window);
	}

	string all = bases_to_indices(ss.str());
	int size = all.size();
	
	//Send sequences to GPU
	cudaStatus = hipMalloc(d_seq, size);
	assert (hipSuccess == cudaStatus);
	hipMemcpy(*d_seq, &all[0], size, hipMemcpyHostToDevice);
	assert (hipSuccess == cudaStatus);
}



//Add Sequences to the Model
void IMM::add(vector<string> & sequences) {
	char *d_seq;
	send_windows_to_gpu(sequences, window, &d_seq);
	
	//invoke counting kernel
	int num_sequences = sequences.size();
	dim3 threads_per_block(num_sequences,window,order+1);
	dim3 blocks(1,1,1);
    counting_kernel<<<blocks, threads_per_block>>>(d_counts, d_seq, order_sum, order, window);

	//Cleanup
	hipFree(d_seq);
}

//The MEAT of the matter: Scoring


//Compute the chi^2 score of a chi2 table
__device__ __host__ float chi_squared_score(int * table, int length) {
	//sum counts in table
	float N = 0.0f;
	for(int i = 0; i < length*2; i++) {
		N += table[i];
	}
	if (N == 0.0f) {
		return 0.0f;
	}
	//Compute Score
	float score = 0.0f;
	for(int i = 0; i < length; i++) {
		float Ri = table[i*2+0] + table[i*2+1];
		for(int j = 0; j < 2; j++) {
			float Cj = 0.0f;
			for(int x = 0; x < length; x++) {
				Cj += table[x*2+j];
			}
			float Eij = Ri*Cj/N;
			float Oij = table[i*2+j];
			if(Eij != 0.0f) {
				score += (Oij - Eij)*(Oij - Eij)/Eij;
			}
		}
	}
	return score;
}


//build two distributions into a chi^2 table
__device__ __host__ void build_chi2_table(int * dist1, int * dist2, int * output, int length) {
	for(int i = 0; i < length; i++) {
		output[i*2] = dist1[i];
		output[i*2+1] = dist2[i];
	}
}

//Test to see if the chi_squared_score and build_chi2_table methods work correctly
bool test_chi_squared_test() {
	int dist1[] = {25,9,11,17};
	int dist2[] = {1,1,59,1};
	int table[8];
	int expected_table[] = {25,1,9,1,11,59,17,1};

	build_chi2_table(dist1, dist2, table, 4);
	float result = chi_squared_score(table, 4);
	int eq = memcmp(table, expected_table, sizeof(table));
	assert(eq == 0);
	return (abs(result - 75.69f) < 0.001);
}


//Build a distribution from an order of the model based on a subsequence
__device__ __host__ void build_distribution(int * model, char * sequence, int length) {
	
}



__global__ void scoring_kernel(int *model, char * sequences, float * scores) {
    int num = threadIdx.x; //sequence number
	int position = threadIdx.y; //position index
	int order = threadIdx.z; //order number

	//Compute lambdas at position based on model

	//Score character based on lambdas
}


void IMM::score(vector<string> & sequences) {
	//send sequences to gpu
	char *d_seq;
	send_windows_to_gpu(sequences, window, &d_seq);

	//Score Positions
	int num_sequences = sequences.size();
	dim3 threads_per_block(num_sequences,2,order+1);
	dim3 blocks(1,1,1);
    counting_kernel<<<blocks, threads_per_block>>>(d_counts, d_seq, order_sum, order, window);

	//Cleanup
	hipFree(d_seq);
}


//Dump model to a vector of ints
void IMM::dump(vector<int> & result) {
	//setup result vector
	result.clear();
	int arr_size = total_bytes / sizeof(int);
	result.resize(arr_size);

	//copy data from gpu
	hipMemcpy(&result[0], d_counts, total_bytes, hipMemcpyDeviceToHost);
}
