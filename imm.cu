#include "hip/hip_runtime.h"
#include "imm.h"

#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <algorithm>
#include <sstream>

using namespace std;

static string bases = "ACGT";
string bases_to_indices(string sequence) {
	string result;
	for(char c : sequence) {
		int index = bases.find(c);
		result.push_back(index);
	}
	return result;
}


IMM::IMM() {
	order = -1;
	window = -1;
	d_counts = NULL;
}

void IMM::dispose() {
	if(d_counts != NULL) {
		hipFree(d_counts);
		d_counts = NULL;
		order = -1;
	}
}


IMM::~IMM() {
	dispose();
}

/* Function to calculate x raised to the power y */
//http://www.geeksforgeeks.org/write-a-c-program-to-calculate-powxn/
int power(int x, unsigned int y)
{
    if( y == 0)
        return 1;
    else if (y%2 == 0)
        return power(x, y/2)*power(x, y/2);
    else
        return x*power(x, y/2)*power(x, y/2);
 
}

//window is the size of the sliding window
//order is the markov-order of the model
void IMM::init(int window, int order) {
	this->order = order;
	this->window = window;

	order_sum = 0;
	for(int i = 0; i <= order; i++) {
		order_sum += power(4, i+1);
	}

	total_bytes = window * order_sum * sizeof(int);
	printf("%d, %d\n", order_sum, total_bytes);
	
    hipError_t cudaStatus;
	cudaStatus = hipMalloc(&d_counts, total_bytes);
	assert (hipSuccess == cudaStatus);
	cudaStatus = hipMemset(d_counts, 0, total_bytes);
	assert (hipSuccess == cudaStatus);
}

__global__ void counting_kernel(int *model, char * sequences, int max_order, int window) {
    int num = threadIdx.x; //sequence number
	int order = threadIdx.y; //order number
    
	//get sequence
	char * sequence = sequences + num * window;

	//get index of sequence
	int index = 0;
	for(int i = 0; i < window; i++) {
		index = index * 4 + *(sequence+i);
	}

	//add to model
	//int model_index = order * window * sizeof(int);
	//TODO: Multiple orders
	model += index;
	atomicAdd(model, 1);
}

//Add Sequences to the Model
void IMM::add(vector<string> sequences) {
    hipError_t cudaStatus;
	//Concatenate sequences
    stringstream ss;
	for(string & seq : sequences) {
		ss << seq.substr(0, window);
	}

    //for_each(sequences.begin(), sequences.end(), [&ss] (const string& s) { ss << s; });
	string all = bases_to_indices(ss.str());
	int size = all.size();
	printf("total_length=%d\n", size);
	
	//Send sequences to GPU
	char *d_seq;
	cudaStatus = hipMalloc(&d_seq, size);
	assert (hipSuccess == cudaStatus);
	hipMemcpy(d_seq, &all[0], size, hipMemcpyHostToDevice);
	assert (hipSuccess == cudaStatus);

	//invoke counting kernel
    counting_kernel<<<1, size>>>(d_counts, d_seq, order, window);

	//Cleanup
	hipFree(d_seq);
}

void IMM::dump(vector<int> & result) {
	//assert(d_counts
	result.clear();
}
