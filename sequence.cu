#include "sequence.h"

#include "hip/hip_runtime.h"
#include ""
#include <assert.h>

Sequence::Sequence() {
	d_sequence = NULL;
	length = 0;
}

Sequence::~Sequence() {
	dispose();
}

void Sequence::load(string sequence) {
    hipError_t cudaStatus;
	length = sequence.length();
    cudaStatus = hipMalloc((void**)&d_sequence, length );
	assert (cudaStatus == hipSuccess);
	cudaStatus = hipMemcpy(d_sequence, &sequence[0], length, hipMemcpyHostToDevice);
	assert (cudaStatus == hipSuccess);
}

void Sequence::dispose() {
	if(d_sequence != NULL) {
		hipFree(d_sequence);
		d_sequence = NULL;
		length = 0;
	}
}
